// RUN: %run_test hipify "%s" "%t" %hipify_args 3 --amap --skip-excluded-preprocessor-conditional-blocks --experimental %clang_args -D__CUDA_API_VERSION_INTERNAL -ferror-limit=500

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
// CHECK: #include "hipfft/hipfftXt.h"
#include "hipfft/hipfftXt.h"
// CHECK-NOT: #include "hipfftXt.h"

int main() {
  printf("25. cufftXt API to hipfftXt API synthetic test\n");

  // CHECK: hipfftXtSubFormat_t fftXtSubFormat_t;
  // CHECK-NEXT: hipfftXtSubFormat_t FFT_XT_FORMAT_INPUT = HIPFFT_XT_FORMAT_INPUT;
  // CHECK-NEXT: hipfftXtSubFormat_t FFT_XT_FORMAT_OUTPUT = HIPFFT_XT_FORMAT_OUTPUT;
  // CHECK-NEXT: hipfftXtSubFormat_t FFT_XT_FORMAT_INPLACE = HIPFFT_XT_FORMAT_INPLACE;
  // CHECK-NEXT: hipfftXtSubFormat_t FFT_XT_FORMAT_INPLACE_SHUFFLED = HIPFFT_XT_FORMAT_INPLACE_SHUFFLED;
  // CHECK-NEXT: hipfftXtSubFormat_t FFT_XT_FORMAT_1D_INPUT_SHUFFLED = HIPFFT_XT_FORMAT_1D_INPUT_SHUFFLED;
  // CHECK-NEXT: hipfftXtSubFormat_t FFT_FORMAT_UNDEFINED = HIPFFT_FORMAT_UNDEFINED;
  hipfftXtSubFormat_t fftXtSubFormat_t;
  hipfftXtSubFormat_t FFT_XT_FORMAT_INPUT = HIPFFT_XT_FORMAT_INPUT;
  hipfftXtSubFormat_t FFT_XT_FORMAT_OUTPUT = HIPFFT_XT_FORMAT_OUTPUT;
  hipfftXtSubFormat_t FFT_XT_FORMAT_INPLACE = HIPFFT_XT_FORMAT_INPLACE;
  hipfftXtSubFormat_t FFT_XT_FORMAT_INPLACE_SHUFFLED = HIPFFT_XT_FORMAT_INPLACE_SHUFFLED;
  hipfftXtSubFormat_t FFT_XT_FORMAT_1D_INPUT_SHUFFLED = HIPFFT_XT_FORMAT_1D_INPUT_SHUFFLED;
  hipfftXtSubFormat_t FFT_FORMAT_UNDEFINED = HIPFFT_FORMAT_UNDEFINED;

  // CHECK: hipfftXtCopyType_t fftXtCopyType_t;
  // CHECK-NEXT: hipfftXtCopyType_t FFT_COPY_HOST_TO_DEVICE = HIPFFT_COPY_HOST_TO_DEVICE;
  // CHECK-NEXT: hipfftXtCopyType_t FFT_COPY_DEVICE_TO_HOST = HIPFFT_COPY_DEVICE_TO_HOST;
  // CHECK-NEXT: hipfftXtCopyType_t FFT_COPY_DEVICE_TO_DEVICE = HIPFFT_COPY_DEVICE_TO_DEVICE;
  // CHECK-NEXT: hipfftXtCopyType_t FFT_COPY_UNDEFINED = HIPFFT_COPY_UNDEFINED;
  hipfftXtCopyType_t fftXtCopyType_t;
  hipfftXtCopyType_t FFT_COPY_HOST_TO_DEVICE = HIPFFT_COPY_HOST_TO_DEVICE;
  hipfftXtCopyType_t FFT_COPY_DEVICE_TO_HOST = HIPFFT_COPY_DEVICE_TO_HOST;
  hipfftXtCopyType_t FFT_COPY_DEVICE_TO_DEVICE = HIPFFT_COPY_DEVICE_TO_DEVICE;
  hipfftXtCopyType_t FFT_COPY_UNDEFINED = HIPFFT_COPY_UNDEFINED;

  // CHECK: hipfftXtCallbackType_t fftXtCallbackType_t;
  // CHECK-NEXT: hipfftXtCallbackType_t FFT_CB_LD_COMPLEX = HIPFFT_CB_LD_COMPLEX;
  // CHECK-NEXT: hipfftXtCallbackType_t FFT_CB_LD_COMPLEX_DOUBLE = HIPFFT_CB_LD_COMPLEX_DOUBLE;
  // CHECK-NEXT: hipfftXtCallbackType_t FFT_CB_LD_REAL = HIPFFT_CB_LD_REAL;
  // CHECK-NEXT: hipfftXtCallbackType_t FFT_CB_LD_REAL_DOUBLE = HIPFFT_CB_LD_REAL_DOUBLE;
  // CHECK-NEXT: hipfftXtCallbackType_t FFT_CB_ST_COMPLEX = HIPFFT_CB_ST_COMPLEX;
  // CHECK-NEXT: hipfftXtCallbackType_t FFT_CB_ST_COMPLEX_DOUBLE = HIPFFT_CB_ST_COMPLEX_DOUBLE;
  // CHECK-NEXT: hipfftXtCallbackType_t FFT_CB_ST_REAL = HIPFFT_CB_ST_REAL;
  // CHECK-NEXT: hipfftXtCallbackType_t FFT_CB_ST_REAL_DOUBLE = HIPFFT_CB_ST_REAL_DOUBLE;
  // CHECK-NEXT: hipfftXtCallbackType_t FFT_CB_UNDEFINED = HIPFFT_CB_UNDEFINED;
  hipfftXtCallbackType_t fftXtCallbackType_t;
  hipfftXtCallbackType_t FFT_CB_LD_COMPLEX = HIPFFT_CB_LD_COMPLEX;
  hipfftXtCallbackType_t FFT_CB_LD_COMPLEX_DOUBLE = HIPFFT_CB_LD_COMPLEX_DOUBLE;
  hipfftXtCallbackType_t FFT_CB_LD_REAL = HIPFFT_CB_LD_REAL;
  hipfftXtCallbackType_t FFT_CB_LD_REAL_DOUBLE = HIPFFT_CB_LD_REAL_DOUBLE;
  hipfftXtCallbackType_t FFT_CB_ST_COMPLEX = HIPFFT_CB_ST_COMPLEX;
  hipfftXtCallbackType_t FFT_CB_ST_COMPLEX_DOUBLE = HIPFFT_CB_ST_COMPLEX_DOUBLE;
  hipfftXtCallbackType_t FFT_CB_ST_REAL = HIPFFT_CB_ST_REAL;
  hipfftXtCallbackType_t FFT_CB_ST_REAL_DOUBLE = HIPFFT_CB_ST_REAL_DOUBLE;
  hipfftXtCallbackType_t FFT_CB_UNDEFINED = HIPFFT_CB_UNDEFINED;

  // CHECK: hipfftResult fftResult;
  hipfftResult fftResult;

  // CHECK: hipfftHandle fftHandle;
  hipfftHandle fftHandle;

  // CHECK: hipfftXtSubFormat fftXtSubFormat;
  hipfftXtSubFormat fftXtSubFormat;

  // CHECK: hipfftXtCopyType fftXtCopyType;
  hipfftXtCopyType fftXtCopyType;

  // CHECK: hipLibXtDesc *descptr = nullptr;
  // CHECK-NEXT: hipLibXtDesc *input_desc = nullptr;
  // CHECK-NEXT: hipLibXtDesc *output_desc = nullptr;
  hipLibXtDesc *descptr = nullptr;
  hipLibXtDesc *input_desc = nullptr;
  hipLibXtDesc *output_desc = nullptr;

  // CHECK: hipLibXtDesc **desc = nullptr;
  hipLibXtDesc **desc = nullptr;

  int *gpu = nullptr;
  int count = 0;
  void *dstptr = nullptr;
  void *srcptr = nullptr;
  int dir = 0;
  int rank = 0;
  long long int *n = nullptr;
  long long int *inembed = nullptr;
  long long int istride = 0;
  long long int idist = 0;
  long long int *onembed = nullptr;
  long long int ostride = 0;
  long long int odist = 0;
  long long int batch = 0;
  size_t *workSize = nullptr;
  void *input = nullptr;
  void *output = nullptr;

  // CHECK: hipDataType executionType;
  // CHECK-NEXT: hipDataType inputType;
  // CHECK-NEXT: hipDataType outputType;
  hipDataType executionType;
  hipDataType inputType;
  hipDataType outputType;

  // CUDA: cufftResult CUFFTAPI cufftXtSetGPUs(cufftHandle handle, int nGPUs, int *whichGPUs);
  // HIP: HIPFFT_EXPORT hipfftResult hipfftXtSetGPUs(hipfftHandle plan, int count, int* gpus);
  // CHECK: fftResult = hipfftXtSetGPUs(fftHandle, count, gpu);
  fftResult = hipfftXtSetGPUs(fftHandle, count, gpu);

  // CUDA: cufftResult CUFFTAPI cufftXtMalloc(cufftHandle plan, cudaLibXtDesc ** descriptor, cufftXtSubFormat format);
  // HIP: HIPFFT_EXPORT hipfftResult hipfftXtMalloc(hipfftHandle plan, hipLibXtDesc** desc, hipfftXtSubFormat format);
  // CHECK: fftResult = hipfftXtMalloc(fftHandle, desc, fftXtSubFormat);
  fftResult = hipfftXtMalloc(fftHandle, desc, fftXtSubFormat);

  // CUDA: cufftResult CUFFTAPI cufftXtMemcpy(cufftHandle plan, void *dstPointer, void *srcPointer, cufftXtCopyType type);
  // HIP: HIPFFT_EXPORT hipfftResult hipfftXtMemcpy(hipfftHandle plan, void* dest, void* src, hipfftXtCopyType type);
  // CHECK: fftResult = hipfftXtMemcpy(fftHandle, dstptr, srcptr, fftXtCopyType);
  fftResult = hipfftXtMemcpy(fftHandle, dstptr, srcptr, fftXtCopyType);

  // CUDA: cufftResult CUFFTAPI cufftXtFree(cudaLibXtDesc *descriptor);
  // HIP: HIPFFT_EXPORT hipfftResult hipfftXtFree(hipLibXtDesc* desc);
  // CHECK: fftResult = hipfftXtFree(descptr);
  fftResult = hipfftXtFree(descptr);

  // CUDA: cufftResult CUFFTAPI cufftXtExecDescriptorC2C(cufftHandle plan, cudaLibXtDesc *input, cudaLibXtDesc *output, int direction);
  // HIP: HIPFFT_EXPORT hipfftResult hipfftXtExecDescriptorC2C(hipfftHandle plan, hipLibXtDesc* input, hipLibXtDesc* output, int direction);
  // CHECK: fftResult = hipfftXtExecDescriptorC2C(fftHandle, input_desc, output_desc, dir);
  fftResult = hipfftXtExecDescriptorC2C(fftHandle, input_desc, output_desc, dir);

  // CUDA: cufftResult CUFFTAPI cufftXtExecDescriptorR2C(cufftHandle plan, cudaLibXtDesc *input, cudaLibXtDesc *output);
  // HIP: HIPFFT_EXPORT hipfftResult hipfftXtExecDescriptorR2C(hipfftHandle plan, hipLibXtDesc* input, hipLibXtDesc* output);
  // CHECK: fftResult = hipfftXtExecDescriptorR2C(fftHandle, input_desc, output_desc);
  fftResult = hipfftXtExecDescriptorR2C(fftHandle, input_desc, output_desc);

  // CUDA: cufftResult CUFFTAPI cufftXtExecDescriptorC2R(cufftHandle plan, cudaLibXtDesc *input, cudaLibXtDesc *output);
  // HIP: HIPFFT_EXPORT hipfftResult hipfftXtExecDescriptorC2R(hipfftHandle plan, hipLibXtDesc* input, hipLibXtDesc* output);
  // CHECK: fftResult = hipfftXtExecDescriptorC2R(fftHandle, input_desc, output_desc);
  fftResult = hipfftXtExecDescriptorC2R(fftHandle, input_desc, output_desc);

  // CUDA: cufftResult CUFFTAPI cufftXtExecDescriptorZ2Z(cufftHandle plan, cudaLibXtDesc *input, cudaLibXtDesc *output, int direction);
  // HIP: HIPFFT_EXPORT hipfftResult hipfftXtExecDescriptorZ2Z(hipfftHandle plan, hipLibXtDesc* input, hipLibXtDesc* output, int direction);
  // CHECK: fftResult = hipfftXtExecDescriptorZ2Z(fftHandle, input_desc, output_desc, dir);
  fftResult = hipfftXtExecDescriptorZ2Z(fftHandle, input_desc, output_desc, dir);

  // CUDA: cufftResult CUFFTAPI cufftXtExecDescriptorD2Z(cufftHandle plan, cudaLibXtDesc *input, cudaLibXtDesc *output);
  // HIP: HIPFFT_EXPORT hipfftResult hipfftXtExecDescriptorD2Z(hipfftHandle plan, hipLibXtDesc* input, hipLibXtDesc* output);
  // CHECK: fftResult = hipfftXtExecDescriptorD2Z(fftHandle, input_desc, output_desc);
  fftResult = hipfftXtExecDescriptorD2Z(fftHandle, input_desc, output_desc);

  // CUDA: cufftResult CUFFTAPI cufftXtExecDescriptorZ2D(cufftHandle plan, cudaLibXtDesc *input, cudaLibXtDesc *output);
  // HIP: HIPFFT_EXPORT hipfftResult hipfftXtExecDescriptorZ2D(hipfftHandle plan, hipLibXtDesc* input, hipLibXtDesc* output);
  // CHECK: fftResult = hipfftXtExecDescriptorZ2D(fftHandle, input_desc, output_desc);
  fftResult = hipfftXtExecDescriptorZ2D(fftHandle, input_desc, output_desc);

#if CUDA_VERSION >= 8000
  // CUDA: cufftResult CUFFTAPI cufftXtMakePlanMany(cufftHandle plan, int rank, long long int *n, long long int *inembed, long long int istride, long long int idist, cudaDataType inputtype, long long int *onembed, long long int ostride, long long int odist, cudaDataType outputtype, long long int batch, size_t *workSize, cudaDataType executiontype);
  // HIP: HIPFFT_EXPORT hipfftResult hipfftXtMakePlanMany(hipfftHandle plan, int rank, long long int* n, long long int* inembed, long long int istride, long long int idist, hipDataType inputType, long long int* onembed, long long int ostride, long long int odist, hipDataType outputType, long long int batch, size_t* workSize, hipDataType executionType);
  // CHECK: fftResult = hipfftXtMakePlanMany(fftHandle, rank, n, inembed, istride, idist, inputType, onembed, ostride, odist, outputType, batch, workSize, executionType);
  fftResult = hipfftXtMakePlanMany(fftHandle, rank, n, inembed, istride, idist, inputType, onembed, ostride, odist, outputType, batch, workSize, executionType);

  // CUDA: cufftResult CUFFTAPI cufftXtGetSizeMany(cufftHandle plan, int rank, long long int *n, long long int *inembed, long long int istride, long long int idist, cudaDataType inputtype, long long int *onembed, long long int ostride, long long int odist, cudaDataType outputtype, long long int batch, size_t *workSize, cudaDataType executiontype);
  // HIP: HIPFFT_EXPORT hipfftResult hipfftXtGetSizeMany(hipfftHandle plan, int rank, long long int* n, long long int* inembed, long long int istride, long long int idist, hipDataType inputType, long long int* onembed, long long int ostride, long long int odist, hipDataType outputType, long long int batch, size_t* workSize, hipDataType executionType);
  // CHECK: fftResult = hipfftXtGetSizeMany(fftHandle, rank, n, inembed, istride, idist, inputType, onembed, ostride, odist, outputType, batch, workSize, executionType);
  fftResult = hipfftXtGetSizeMany(fftHandle, rank, n, inembed, istride, idist, inputType, onembed, ostride, odist, outputType, batch, workSize, executionType);

  // CUDA: cufftResult CUFFTAPI cufftXtExec(cufftHandle plan, void *input, void *output, int direction);
  // HIP: HIPFFT_EXPORT hipfftResult hipfftXtExec(hipfftHandle plan, void* input, void* output, int direction);
  // CHECK: fftResult = hipfftXtExec(fftHandle, input, output, dir);
  fftResult = hipfftXtExec(fftHandle, input, output, dir);

  // CUDA: cufftResult CUFFTAPI cufftXtExecDescriptor(cufftHandle plan, cudaLibXtDesc *input, cudaLibXtDesc *output, int direction);
  // HIP: HIPFFT_EXPORT hipfftResult hipfftXtExecDescriptor(hipfftHandle plan, hipLibXtDesc* input, hipLibXtDesc* output, int direction);
  // CHECK: fftResult = hipfftXtExecDescriptor(fftHandle, input_desc, output_desc, dir);
  fftResult = hipfftXtExecDescriptor(fftHandle, input_desc, output_desc, dir);
#endif

  return 0;
}
