// RUN: %run_test hipify "%s" "%t" %hipify_args 3 --amap --skip-excluded-preprocessor-conditional-blocks --experimental %clang_args -D__CUDA_API_VERSION_INTERNAL -ferror-limit=500

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
// CHECK: #include "hipfft/hipfftXt.h"
#include "hipfft/hipfftXt.h"
// CHECK-NOT: #include "hipfftXt.h"

int main() {
  printf("25. cufftXt API to hipfftXt API synthetic test\n");

  // CHECK: hipfftXtSubFormat_t fftXtSubFormat_t;
  // CHECK-NEXT: hipfftXtSubFormat_t FFT_XT_FORMAT_INPUT = HIPFFT_XT_FORMAT_INPUT;
  // CHECK-NEXT: hipfftXtSubFormat_t FFT_XT_FORMAT_OUTPUT = HIPFFT_XT_FORMAT_OUTPUT;
  // CHECK-NEXT: hipfftXtSubFormat_t FFT_XT_FORMAT_INPLACE = HIPFFT_XT_FORMAT_INPLACE;
  // CHECK-NEXT: hipfftXtSubFormat_t FFT_XT_FORMAT_INPLACE_SHUFFLED = HIPFFT_XT_FORMAT_INPLACE_SHUFFLED;
  // CHECK-NEXT: hipfftXtSubFormat_t FFT_XT_FORMAT_1D_INPUT_SHUFFLED = HIPFFT_XT_FORMAT_1D_INPUT_SHUFFLED;
  // CHECK-NEXT: hipfftXtSubFormat_t FFT_FORMAT_UNDEFINED = HIPFFT_FORMAT_UNDEFINED;
  hipfftXtSubFormat_t fftXtSubFormat_t;
  hipfftXtSubFormat_t FFT_XT_FORMAT_INPUT = HIPFFT_XT_FORMAT_INPUT;
  hipfftXtSubFormat_t FFT_XT_FORMAT_OUTPUT = HIPFFT_XT_FORMAT_OUTPUT;
  hipfftXtSubFormat_t FFT_XT_FORMAT_INPLACE = HIPFFT_XT_FORMAT_INPLACE;
  hipfftXtSubFormat_t FFT_XT_FORMAT_INPLACE_SHUFFLED = HIPFFT_XT_FORMAT_INPLACE_SHUFFLED;
  hipfftXtSubFormat_t FFT_XT_FORMAT_1D_INPUT_SHUFFLED = HIPFFT_XT_FORMAT_1D_INPUT_SHUFFLED;
  hipfftXtSubFormat_t FFT_FORMAT_UNDEFINED = HIPFFT_FORMAT_UNDEFINED;

  // CHECK: hipfftXtCopyType_t fftXtCopyType_t;
  // CHECK-NEXT: hipfftXtCopyType_t FFT_COPY_HOST_TO_DEVICE = HIPFFT_COPY_HOST_TO_DEVICE;
  // CHECK-NEXT: hipfftXtCopyType_t FFT_COPY_DEVICE_TO_HOST = HIPFFT_COPY_DEVICE_TO_HOST;
  // CHECK-NEXT: hipfftXtCopyType_t FFT_COPY_DEVICE_TO_DEVICE = HIPFFT_COPY_DEVICE_TO_DEVICE;
  // CHECK-NEXT: hipfftXtCopyType_t FFT_COPY_UNDEFINED = HIPFFT_COPY_UNDEFINED;
  hipfftXtCopyType_t fftXtCopyType_t;
  hipfftXtCopyType_t FFT_COPY_HOST_TO_DEVICE = HIPFFT_COPY_HOST_TO_DEVICE;
  hipfftXtCopyType_t FFT_COPY_DEVICE_TO_HOST = HIPFFT_COPY_DEVICE_TO_HOST;
  hipfftXtCopyType_t FFT_COPY_DEVICE_TO_DEVICE = HIPFFT_COPY_DEVICE_TO_DEVICE;
  hipfftXtCopyType_t FFT_COPY_UNDEFINED = HIPFFT_COPY_UNDEFINED;

  // CHECK: hipfftXtCallbackType_t fftXtCallbackType_t;
  // CHECK-NEXT: hipfftXtCallbackType_t FFT_CB_LD_COMPLEX = HIPFFT_CB_LD_COMPLEX;
  // CHECK-NEXT: hipfftXtCallbackType_t FFT_CB_LD_COMPLEX_DOUBLE = HIPFFT_CB_LD_COMPLEX_DOUBLE;
  // CHECK-NEXT: hipfftXtCallbackType_t FFT_CB_LD_REAL = HIPFFT_CB_LD_REAL;
  // CHECK-NEXT: hipfftXtCallbackType_t FFT_CB_LD_REAL_DOUBLE = HIPFFT_CB_LD_REAL_DOUBLE;
  // CHECK-NEXT: hipfftXtCallbackType_t FFT_CB_ST_COMPLEX = HIPFFT_CB_ST_COMPLEX;
  // CHECK-NEXT: hipfftXtCallbackType_t FFT_CB_ST_COMPLEX_DOUBLE = HIPFFT_CB_ST_COMPLEX_DOUBLE;
  // CHECK-NEXT: hipfftXtCallbackType_t FFT_CB_ST_REAL = HIPFFT_CB_ST_REAL;
  // CHECK-NEXT: hipfftXtCallbackType_t FFT_CB_ST_REAL_DOUBLE = HIPFFT_CB_ST_REAL_DOUBLE;
  // CHECK-NEXT: hipfftXtCallbackType_t FFT_CB_UNDEFINED = HIPFFT_CB_UNDEFINED;
  hipfftXtCallbackType_t fftXtCallbackType_t;
  hipfftXtCallbackType_t FFT_CB_LD_COMPLEX = HIPFFT_CB_LD_COMPLEX;
  hipfftXtCallbackType_t FFT_CB_LD_COMPLEX_DOUBLE = HIPFFT_CB_LD_COMPLEX_DOUBLE;
  hipfftXtCallbackType_t FFT_CB_LD_REAL = HIPFFT_CB_LD_REAL;
  hipfftXtCallbackType_t FFT_CB_LD_REAL_DOUBLE = HIPFFT_CB_LD_REAL_DOUBLE;
  hipfftXtCallbackType_t FFT_CB_ST_COMPLEX = HIPFFT_CB_ST_COMPLEX;
  hipfftXtCallbackType_t FFT_CB_ST_COMPLEX_DOUBLE = HIPFFT_CB_ST_COMPLEX_DOUBLE;
  hipfftXtCallbackType_t FFT_CB_ST_REAL = HIPFFT_CB_ST_REAL;
  hipfftXtCallbackType_t FFT_CB_ST_REAL_DOUBLE = HIPFFT_CB_ST_REAL_DOUBLE;
  hipfftXtCallbackType_t FFT_CB_UNDEFINED = HIPFFT_CB_UNDEFINED;
  return 0;
}
