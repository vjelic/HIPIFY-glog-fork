// RUN: %run_test hipify "%s" "%t" %hipify_args 3 --amap --skip-excluded-preprocessor-conditional-blocks --experimental %clang_args -ferror-limit=500

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
// CHECK: #include "hip/hip_fp8.h"
#include "hip/hip_fp8.h"
// CHECK-NOT: #include "hip/hip_fp8.h"
// CHECK-NOT: #include "cuda_fp8.h"

int main() {
  printf("24.before_11080_after_12011, CUDA Device API to HIP Device API synthetic test\n");

  double da = 0.0f;
  double dx = 0.0f;
  float fa = 0.0f;
  float fx = 0.0f;
  double2 d2 = { 0.0f, 0.0f };
  float2 f2 = { 0.0f, 0.0f };
  __half_raw hrx = { 0 };
  __half2_raw h2rx = { 0, 0 };

#if CUDA_VERSION >= 11000
  // CHECK: __hip_bfloat16 bf16 = { 0 };
  __hip_bfloat16 bf16 = { 0 };
  // CHECK: __hip_bfloat162 bf162 = { 0, 0 };
  // CHECK-NEXT: __hip_bfloat162 bf162a = { 0, 0 };
  // CHECK-NEXT: __hip_bfloat162 bf162b = { 0, 0 };
  __hip_bfloat162 bf162 = { 0, 0 };
  __hip_bfloat162 bf162a = { 0, 0 };
  __hip_bfloat162 bf162b = { 0, 0 };

#if CUDA_VERSION < 11080 || CUDA_VERSION >= 12000
  // CUDA: __CUDA_HOSTDEVICE_BF16_DECL__ __nv_bfloat162 __bfloat162bfloat162(const __nv_bfloat16 a);
  // HIP: __BF16_HOST_DEVICE_STATIC__ __hip_bfloat162 __bfloat162bfloat162(const __hip_bfloat16 a);
  // CHECK: bf162 = __bfloat162bfloat162(bf16);
  bf162 = __bfloat162bfloat162(bf16);

  // CUDA: __CUDA_HOSTDEVICE_BF16_DECL__ __nv_bfloat162 __lows2bfloat162(const __nv_bfloat162 a, const __nv_bfloat162 b);
  // HIP: __BF16_HOST_DEVICE_STATIC__ __hip_bfloat162 __lows2bfloat162(const __hip_bfloat162 a, const __hip_bfloat162 b);
  // CHECK: bf162 = __lows2bfloat162(bf162a, bf162b);
  bf162 = __lows2bfloat162(bf162a, bf162b);

  // CUDA: __CUDA_HOSTDEVICE_BF16_DECL__ __nv_bfloat162 __highs2bfloat162(const __nv_bfloat162 a, const __nv_bfloat162 b);
  // HIP: __BF16_HOST_DEVICE_STATIC__ __hip_bfloat162 __highs2bfloat162(const __hip_bfloat162 a, const __hip_bfloat162 b);
  // CHECK: bf162 = __highs2bfloat162(bf162a, bf162b);
  bf162 = __highs2bfloat162(bf162a, bf162b);

  // CUDA: __CUDA_HOSTDEVICE_BF16_DECL__ __nv_bfloat16 __high2bfloat16(const __nv_bfloat162 a);
  // HIP: __BF16_HOST_DEVICE_STATIC__ __hip_bfloat16 __high2bfloat16(const __hip_bfloat162 a);
  // CHECK: bf16 = __high2bfloat16(bf162a);
  bf16 = __high2bfloat16(bf162a);
#endif

#endif

  return 0;
}
