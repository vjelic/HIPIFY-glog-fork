#include "hip/hip_runtime.h"
// RUN: %run_test hipify "%s" "%t" %hipify_args 3 --amap --skip-excluded-preprocessor-conditional-blocks --experimental %clang_args -D__CUDA_API_VERSION_INTERNAL -ferror-limit=500

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
// CHECK: #include "hiptensor.h"
#include "hiptensor.h"
// CHECK-NOT: #include "hiptensor.h"

int main() {
  printf("25. cuTensor API to hipTensor API synthetic test\n");

  // CHECK: hiptensorHandle_t handle;
  // CHECK-NEXT: const hiptensorHandle_t *handle_c = nullptr;
  // CHECK-NEXT: hiptensorHandle_t *handle2 = nullptr;
  hiptensorHandle_t handle;
  const hiptensorHandle_t *handle_c = nullptr;
  hiptensorHandle_t *handle2 = nullptr;

  //CHECK: hiptensorStatus_t status;
  hiptensorStatus_t status;

  //CHECK: hiptensorTensorDescriptor_t *tensorDescriptor = nullptr;
  //CHECK-NEXT: hiptensorTensorDescriptor_t *descA = nullptr;
  //CHECK-NEXT: hiptensorTensorDescriptor_t *descB = nullptr;
  //CHECK-NEXT: hiptensorTensorDescriptor_t *descC = nullptr;
  //CHECK-NEXT: hiptensorTensorDescriptor_t *descD = nullptr;
  hiptensorTensorDescriptor_t *tensorDescriptor = nullptr;
  hiptensorTensorDescriptor_t *descA = nullptr;
  hiptensorTensorDescriptor_t *descB = nullptr;
  hiptensorTensorDescriptor_t *descC = nullptr;
  hiptensorTensorDescriptor_t *descD = nullptr;

#if CUDA_VERSION >= 8000
  // CHECK: hipDataType dataType;
  hipDataType dataType;
#endif

  // CHECK: hipStream_t stream_t;
  hipStream_t stream_t;

  const uint32_t numModes = 0;
  const int64_t *extent = nullptr;
  const int64_t *stride = nullptr;
  const uint64_t workspaceSize = 0;
  uint64_t workspaceSize2 = 0;
  const void *alpha = nullptr;
  const void *A = nullptr;
  const int32_t *modeA = nullptr;
  void *B = nullptr;
  const void *B_1 = nullptr;
  const void *beta = nullptr;
  const int32_t *modeB = nullptr;
  const void *C = nullptr;
  const int32_t *modeC = nullptr;
  void *D = nullptr;
  const int32_t *modeD = nullptr;
  void *workspace = nullptr;
  const char *err = nullptr;
  const char *log = nullptr;
  size_t ver = 0;
  FILE *file = nullptr;
  int32_t level = 0;
  int32_t mask = 0;

#if CUTENSOR_MAJOR >= 2
  // CHECK: hiptensorComputeType_t tensorDataType_t;
  // CHECK-NEXT hiptensorComputeType_t TENSOR_R_16F = HIPTENSOR_COMPUTE_16F;
  // CHECK-NEXT hiptensorComputeType_t TENSOR_R_16BF = HIPTENSOR_COMPUTE_16BF;
  // CHECK-NEXT hiptensorComputeType_t TENSOR_R_32F = HIPTENSOR_COMPUTE_32F;
  // CHECK-NEXT hiptensorComputeType_t TENSOR_C_32F = HIPTENSOR_COMPUTE_C32F;
  // CHECK-NEXT hiptensorComputeType_t TENSOR_R_64F = HIPTENSOR_COMPUTE_64F;
  // CHECK-NEXT hiptensorComputeType_t TENSOR_C_64F = HIPTENSOR_COMPUTE_C64F;
  // CHECK-NEXT hiptensorComputeType_t TENSOR_R_8I = HIPTENSOR_COMPUTE_8I;
  // CHECK-NEXT hiptensorComputeType_t TENSOR_R_8U = HIPTENSOR_COMPUTE_8U;
  // CHECK-NEXT hiptensorComputeType_t TENSOR_R_32I = HIPTENSOR_COMPUTE_32I;
  // CHECK-NEXT hiptensorComputeType_t TENSOR_R_32U = HIPTENSOR_COMPUTE_32U;
  hiptensorDataType_t tensorDataType_t;
  hiptensorDataType_t TENSOR_R_16F = HIPTENSOR_R_16F;
  hiptensorDataType_t TENSOR_R_16BF = HIPTENSOR_R_16BF;
  hiptensorDataType_t TENSOR_R_32F = HIPTENSOR_R_32F;
  hiptensorDataType_t TENSOR_C_32F = HIPTENSOR_C_32F;
  hiptensorDataType_t TENSOR_R_64F = HIPTENSOR_R_64F;
  hiptensorDataType_t TENSOR_C_64F = HIPTENSOR_C_64F;
  hiptensorDataType_t TENSOR_R_8I = HIPTENSOR_R_8I;
  hiptensorDataType_t TENSOR_R_8U = HIPTENSOR_R_8U;
  hiptensorDataType_t TENSOR_R_32I = HIPTENSOR_R_32I;
  hiptensorDataType_t TENSOR_R_32U = HIPTENSOR_R_32U;

  // CHECK: hiptensorContractionPlan_t tensorPlan2_t;
  hiptensorPlan_t tensorPlan2_t;

  // CUDA: hiptensorStatus_t hiptensorContract(const hiptensorHandle_t handle, const hiptensorPlan_t plan, const void* alpha, const void *A, const void *B, const void* beta, const void *C, void *D, void* workspace, uint64_t workspaceSize, hipStream_t stream);
  // HIP: hiptensorStatus_t hiptensorContraction(const hiptensorHandle_t* handle, const hiptensorContractionPlan_t* plan, const void* alpha, const void* A, const void* B, const void* beta, const void* C, void* D, void* workspace, uint64_t workspaceSize, hipStream_t stream);
  // CHECK: status = hiptensorContraction(handle, tensorPlan2_t, alpha, A, B_1, beta, C, D, workspace,  workspaceSize2, stream_t);
  status = hiptensorContract(handle, tensorPlan2_t, alpha, A, B_1, beta, C, D, workspace, workspaceSize2, stream_t);

   // CUDA: hiptensorStatus_t hiptensorCreate(hiptensorHandle_t* handle);
   // HIP: hiptensorStatus_t hiptensorCreate(hiptensorHandle_t** handle);
   // CHECK: status = hiptensorCreate(&handle);
   status = hiptensorCreate(&handle);

   // CUDA: hiptensorStatus_t hiptensorDestroy(hiptensorHandle_t handle);
   // HIP: hiptensorStatus_t hiptensorDestroy(hiptensorHandle_t* handle);
   // CHECK: status = hiptensorDestroy(handle);
   status = hiptensorDestroy(handle);
#endif

#if CUTENSOR_MAJOR >= 1
  // CHECK: hiptensorOperator_t tensorOperator_t;
  // CHECK-NEXT hiptensorOperator_t TENSOR_OP_IDENTITY = HIPTENSOR_OP_IDENTITY;
  // CHECK-NEXT hiptensorOperator_t TENSOR_OP_SQRT = HIPTENSOR_OP_SQRT;
  // CHECK-NEXT hiptensorOperator_t TENSOR_OP_ADD = HIPTENSOR_OP_ADD;
  // CHECK-NEXT hiptensorOperator_t TENSOR_OP_MUL = HIPTENSOR_OP_MUL;
  // CHECK-NEXT hiptensorOperator_t TENSOR_OP_MAX = HIPTENSOR_OP_MAX;
  // CHECK-NEXT hiptensorOperator_t TENSOR_OP_MIN = HIPTENSOR_OP_MIN;
  // CHECK-NEXT hiptensorOperator_t TENSOR_OP_UNKNOWN = HIPTENSOR_OP_UNKNOWN;
  hiptensorOperator_t tensorOperator_t;
  hiptensorOperator_t TENSOR_OP_IDENTITY = HIPTENSOR_OP_IDENTITY;
  hiptensorOperator_t TENSOR_OP_SQRT = HIPTENSOR_OP_SQRT;
  hiptensorOperator_t TENSOR_OP_ADD = HIPTENSOR_OP_ADD;
  hiptensorOperator_t TENSOR_OP_MUL = HIPTENSOR_OP_MUL;
  hiptensorOperator_t TENSOR_OP_MAX = HIPTENSOR_OP_MAX;
  hiptensorOperator_t TENSOR_OP_MIN = HIPTENSOR_OP_MIN;
  hiptensorOperator_t TENSOR_OP_UNKNOWN = HIPTENSOR_OP_UNKNOWN;

  // CHECK: hiptensorStatus_t tensorStatus_t;
  // CHECK-NEXT hiptensorStatus_t TENSOR_STATUS_SUCCESS = HIPTENSOR_STATUS_SUCCESS;
  // CHECK-NEXT hiptensorStatus_t TENSOR_STATUS_NOT_INITIALIZED = HIPTENSOR_STATUS_NOT_INITIALIZED;
  // CHECK-NEXT hiptensorStatus_t TENSOR_STATUS_ALLOC_FAILED = HIPTENSOR_STATUS_ALLOC_FAILED;
  // CHECK-NEXT hiptensorStatus_t TENSOR_STATUS_INVALID_VALUE = HIPTENSOR_STATUS_INVALID_VALUE;
  // CHECK-NEXT hiptensorStatus_t TENSOR_STATUS_ARCH_MISMATCH = HIPTENSOR_STATUS_ARCH_MISMATCH;
  // CHECK-NEXT hiptensorStatus_t TENSOR_STATUS_EXECUTION_FAILED = HIPTENSOR_STATUS_EXECUTION_FAILED;
  // CHECK-NEXT hiptensorStatus_t TENSOR_STATUS_INTERNAL_ERROR = HIPTENSOR_STATUS_INTERNAL_ERROR;
  // CHECK-NEXT hiptensorStatus_t TENSOR_STATUS_NOT_SUPPORTED = HIPTENSOR_STATUS_NOT_SUPPORTED;
  // CHECK-NEXT hiptensorStatus_t TENSOR_STATUS_INSUFFICIENT_WORKSPACE = HIPTENSOR_STATUS_INSUFFICIENT_WORKSPACE;
  // CHECK-NEXT hiptensorStatus_t TENSOR_STATUS_INSUFFICIENT_DRIVER = HIPTENSOR_STATUS_INSUFFICIENT_DRIVER;
  hiptensorStatus_t tensorStatus_t;
  hiptensorStatus_t TENSOR_STATUS_SUCCESS = HIPTENSOR_STATUS_SUCCESS;
  hiptensorStatus_t TENSOR_STATUS_NOT_INITIALIZED = HIPTENSOR_STATUS_NOT_INITIALIZED;
  hiptensorStatus_t TENSOR_STATUS_ALLOC_FAILED = HIPTENSOR_STATUS_ALLOC_FAILED;
  hiptensorStatus_t TENSOR_STATUS_INVALID_VALUE = HIPTENSOR_STATUS_INVALID_VALUE;
  hiptensorStatus_t TENSOR_STATUS_ARCH_MISMATCH = HIPTENSOR_STATUS_ARCH_MISMATCH;
  hiptensorStatus_t TENSOR_STATUS_EXECUTION_FAILED = HIPTENSOR_STATUS_EXECUTION_FAILED;
  hiptensorStatus_t TENSOR_STATUS_INTERNAL_ERROR = HIPTENSOR_STATUS_INTERNAL_ERROR;
  hiptensorStatus_t TENSOR_STATUS_NOT_SUPPORTED = HIPTENSOR_STATUS_NOT_SUPPORTED;
  hiptensorStatus_t TENSOR_STATUS_INSUFFICIENT_WORKSPACE = HIPTENSOR_STATUS_INSUFFICIENT_WORKSPACE;
  hiptensorStatus_t TENSOR_STATUS_INSUFFICIENT_DRIVER = HIPTENSOR_STATUS_INSUFFICIENT_DRIVER;

  // CHECK: hiptensorAlgo_t tensorAlgo_t;
  // CHECK-NEXT hiptensorAlgo_t TENSOR_ALGO_DEFAULT = HIPTENSOR_ALGO_DEFAULT;
  hiptensorAlgo_t tensorAlgo_t;
  hiptensorAlgo_t TENSOR_ALGO_DEFAULT = HIPTENSOR_ALGO_DEFAULT;

  // CHECK: hiptensorWorksizePreference_t tensorWorksizePreference_t;
  // Check-NEXT TENSOR_WORKSPACE_MIN = HIPTENSOR_WORKSPACE_MIN;
  // CHECK-NEXT TENSOR_WORKSPACE_MAX = HIPTENSOR_WORKSPACE_MAX;
  hiptensorWorksizePreference_t tensorWorksizePreference_t;
  hiptensorWorksizePreference_t TENSOR_WORKSPACE_MIN = HIPTENSOR_WORKSPACE_MIN;
  hiptensorWorksizePreference_t TENSOR_WORKSPACE_MAX = HIPTENSOR_WORKSPACE_MAX;

  // CUDA: const char* hiptensorGetErrorString(const hiptensorStatus_t error);
  // HIP: const char* hiptensorGetErrorString(const hiptensorStatus_t error);
  // CHECK: err = hiptensorGetErrorString(status);
  err = hiptensorGetErrorString(status);

  // CUDA: size_t hiptensorGetHiprtVersion();
  // HIP: int hiptensorGetHiprtVersion();
  // CHECK: ver = hiptensorGetHiprtVersion();
  ver = hiptensorGetHiprtVersion();
#endif

#if (CUTENSOR_MAJOR == 1 && CUTENSOR_MINOR >= 4) || CUTENSOR_MAJOR >= 2
  // CHECK: hiptensorAlgo_t TENSOR_ALGO_DEFAULT_PATIENT = HIPTENSOR_ALGO_DEFAULT_PATIENT;
  hiptensorAlgo_t TENSOR_ALGO_DEFAULT_PATIENT = HIPTENSOR_ALGO_DEFAULT_PATIENT;
#endif

#if (CUTENSOR_MAJOR >= 1 && CUTENSOR_MAJOR < 2)
  // CHECK: hiptensorComputeType_t tensorComputeType_t;
  hiptensorComputeDescriptor_t tensorComputeType_t;

#if CUTENSOR_MINOR >= 2
  // CHECK: hiptensorStatus_t TENSOR_STATUS_IO_ERROR = HIPTENSOR_STATUS_IO_ERROR;
  hiptensorStatus_t TENSOR_STATUS_IO_ERROR = HIPTENSOR_STATUS_IO_ERROR;

  // CHECK hiptensorComputeType_t TENSOR_COMPUTE_16F = HIPTENSOR_COMPUTE_16F;
  // CHECK-NEXT hiptensorComputeType_t TENSOR_COMPUTE_16BF = HIPTENSOR_COMPUTE_16BF;
  // CHECK-NEXT hiptensorComputeType_t TENSOR_COMPUTE_32F = HIPTENSOR_COMPUTE_32F;
  // CHECK-NEXT hiptensorComputeType_t TENSOR_COMPUTE_64F = HIPTENSOR_COMPUTE_64F;
  // CHECK-NEXT hiptensorComputeType_t TENSOR_COMPUTE_8U = HIPTENSOR_COMPUTE_8U;
  // CHECK-NEXT hiptensorComputeType_t TENSOR_COMPUTE_8I = HIPTENSOR_COMPUTE_8I;
  // CHECK-NEXT hiptensorComputeType_t TENSOR_COMPUTE_32U = HIPTENSOR_COMPUTE_32U;
  // CHECK-NEXT hiptensorComputeType_t TENSOR_COMPUTE_32I = HIPTENSOR_COMPUTE_32I;
  hiptensorComputeDescriptor_t TENSOR_COMPUTE_16F = HIPTENSOR_COMPUTE_DESC_16F;
  hiptensorComputeDescriptor_t TENSOR_COMPUTE_16BF = HIPTENSOR_COMPUTE_DESC_16BF;
  hiptensorComputeDescriptor_t TENSOR_COMPUTE_32F = HIPTENSOR_COMPUTE_DESC_32F;
  hiptensorComputeDescriptor_t TENSOR_COMPUTE_64F = HIPTENSOR_COMPUTE_DESC_64F;
  hiptensorComputeDescriptor_t TENSOR_COMPUTE_8U = HIPTENSOR_COMPUTE_DESC_8U;
  hiptensorComputeDescriptor_t TENSOR_COMPUTE_8I = HIPTENSOR_COMPUTE_DESC_8I;
  hiptensorComputeDescriptor_t TENSOR_COMPUTE_32U = HIPTENSOR_COMPUTE_DESC_32U;
  hiptensorComputeDescriptor_t TENSOR_COMPUTE_32I = HIPTENSOR_COMPUTE_DESC_32I;
#endif

  // CHECK: const hiptensorContractionPlan_t *plan_c = nullptr;
  const hiptensorContractionPlan_t *plan_c = nullptr;

  // CHECK: hiptensorWorksizePreference_t TENSOR_WORKSPACE_RECOMMENDED = HIPTENSOR_WORKSPACE_RECOMMENDED;
  hiptensorWorksizePreference_t TENSOR_WORKSPACE_RECOMMENDED = CUTENSOR_WORKSPACE_RECOMMENDED;

#if CUDA_VERSION >= 8000
  // CUDA: hiptensorStatus_t hiptensorInitTensorDescriptor(const hiptensorHandle_t* handle, hiptensorTensorDescriptor_t* desc, const uint32_t numModes, const int64_t extent[], const int64_t stride[], hipDataType dataType, hiptensorOperator_t unaryOp);
  // HIP: hiptensorStatus_t hiptensorInitTensorDescriptor(const hiptensorHandle_t* handle, hiptensorTensorDescriptor_t* desc, const uint32_t numModes, const int64_t lens[], const int64_t strides[], hipDataType dataType, hiptensorOperator_t unaryOp);
  // CHECK: status = hiptensorInitTensorDescriptor(handle_c, tensorDescriptor, numModes, extent, stride, dataType, tensorOperator_t);
  status = hiptensorInitTensorDescriptor(handle_c, tensorDescriptor, numModes, extent, stride, dataType, tensorOperator_t);
#endif

  // CUDA: hiptensorStatus_t hiptensorPermutation(const hiptensorHandle_t* handle, const void* alpha, const void* A, const hiptensorTensorDescriptor_t* descA, const int32_t modeA[], void* B, const hiptensorTensorDescriptor_t* descB, const int32_t modeB[], const hipDataType typeScalar, const hipStream_t stream);
  // HIP: hiptensorStatus_t hiptensorPermutation(const hiptensorHandle_t* handle, const void* alpha, const void* A, const hiptensorTensorDescriptor_t* descA, const int32_t modeA[], void* B, const hiptensorTensorDescriptor_t* descB, const int32_t modeB[], const hipDataType typeScalar, const hipStream_t stream);
  // CHECK: status = hiptensorPermutation(handle_c, alpha, A, descA, modeA, B, descB, modeB, dataType, stream_t);
  status = hiptensorPermutation(handle_c, alpha, A, descA, modeA, B, descB, modeB, dataType, stream_t);

  // CUDA: hiptensorStatus_t hiptensorContraction(const hiptensorHandle_t* handle, const hiptensorContractionPlan_t* plan, const void* alpha, const void* A, const void* B, const void* beta, const void* C, void* D, void *workspace, uint64_t workspaceSize, hipStream_t stream);
  // HIP: hiptensorStatus_t hiptensorContraction(const hiptensorHandle_t* handle, const hiptensorContractionPlan_t* plan, const void* alpha, const void* A, const void* B, const void* beta, const void* C, void* D, void* workspace, uint64_t workspaceSize, hipStream_t stream);
  // CHECK: status = hiptensorContraction(handle_c, plan_c, alpha, A, B_1, beta, C, D, workspace, workspaceSize, stream_t);
  status = hiptensorContraction(handle_c, plan_c, alpha, A, B_1, beta, C, D, workspace, workspaceSize, stream_t);

  // CUDA: hiptensorStatus_t hiptensorReduction(const hiptensorHandle_t* handle, const void* alpha, const void* A, const hiptensorTensorDescriptor_t* descA, const int32_t modeA[], const void* beta, const void* C, const hiptensorTensorDescriptor_t* descC, const int32_t modeC[], void* D, const hiptensorTensorDescriptor_t* descD, const int32_t modeD[], hiptensorOperator_t opReduce, hiptensorComputeDescriptor_t typeCompute, void *workspace, uint64_t workspaceSize, hipStream_t stream);
  // HIP: hiptensorStatus_t hiptensorReduction(const hiptensorHandle_t* handle, const void* alpha, const void* A, const hiptensorTensorDescriptor_t* descA, const int32_t modeA[], const void* beta, const void* C, const hiptensorTensorDescriptor_t* descC, const int32_t modeC[], void* D, const hiptensorTensorDescriptor_t* descD,  const int32_t modeD[], hiptensorOperator_t opReduce, hiptensorComputeType_t typeCompute, void* workspace, uint64_t workspaceSize, hipStream_t stream);
  // CHECK: status = hiptensorReduction(handle_c, alpha, A, descA, modeA, beta, C, descC, modeC, D, descD, modeD, tensorOperator_t, tensorComputeType_t, workspace, workspaceSize2, stream_t);
  status = hiptensorReduction(handle_c, alpha, A, descA, modeA, beta, C, descC, modeC, D, descD, modeD, tensorOperator_t, tensorComputeType_t, workspace, workspaceSize2, stream_t);
#endif

#if (CUTENSOR_MAJOR == 1 && CUTENSOR_MINOR >= 7)
  // CUDA: hiptensorStatus_t hiptensorCreate(hiptensorHandle_t* handle);
  // HIP: hiptensorStatus_t hiptensorCreate(hiptensorHandle_t** handle);
  // CHECK: status = hiptensorCreate(&handle2);
  status = hiptensorCreate(&handle2);

  // CUDA: hiptensorStatus_t hiptensorDestroy(hiptensorHandle_t handle);
  // HIP: hiptensorStatus_t hiptensorDestroy(hiptensorHandle_t* handle);
  // CHECK: status = hiptensorDestroy(handle2);
  status = hiptensorDestroy(handle2);
 #endif

#if (CUTENSOR_MAJOR == 1 && CUTENSOR_MINOR >= 3 && CUTENSOR_PATCH >= 2) || CUTENSOR_MAJOR >= 2
  // CHECK: hiptensorLoggerCallback_t callback;
  hiptensorLoggerCallback_t callback;

  // CUDA: hiptensorStatus_t hiptensorLoggerSetCallback(hiptensorLoggerCallback_t callback);
  // HIP: hiptensorStatus_t hiptensorLoggerSetCallback(hiptensorLoggerCallback_t callback);
  // CHECK: status = hiptensorLoggerSetCallback(callback);
  status = hiptensorLoggerSetCallback(callback);

  // CUDA: hiptensorStatus_t hiptensorLoggerSetFile(FILE* file);
  // HIP: hiptensorStatus_t hiptensorLoggerSetFile(FILE* file);
  // CHECK: status = hiptensorLoggerSetFile(file);
  status = hiptensorLoggerSetFile(file);

  // CUDA: hiptensorStatus_t hiptensorLoggerOpenFile(const char* logFile);
  // HIP: hiptensorStatus_t hiptensorLoggerOpenFile(const char* logFile);
  // CHECK: status = hiptensorLoggerOpenFile(log);
  status = hiptensorLoggerOpenFile(log);

  // CUDA: hiptensorStatus_t hiptensorLoggerSetLevel(int32_t level);
  // HIP: hiptensorStatus_t hiptensorLoggerSetLevel(hiptensorLogLevel_t level);
  // CHECK: status = hiptensorLoggerSetLevel(level);
  status = hiptensorLoggerSetLevel(level);

  // CUDA: hiptensorStatus_t hiptensorLoggerSetMask(int32_t mask);
  // HIP: hiptensorStatus_t hiptensorLoggerSetMask(int32_t mask);
  // CHECK: status = hiptensorLoggerSetMask(mask);
  status = hiptensorLoggerSetMask(mask);

  // CUDA: hiptensorStatus_t hiptensorLoggerForceDisable();
  // HIP: hiptensorStatus_t hiptensorLoggerForceDisable();
  // CHECK: status = hiptensorLoggerForceDisable();
  status = hiptensorLoggerForceDisable();
#endif

  return 0;
}
