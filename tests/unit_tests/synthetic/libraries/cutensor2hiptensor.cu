#include "hip/hip_runtime.h"
// RUN: %run_test hipify "%s" "%t" %hipify_args 3 --amap --skip-excluded-preprocessor-conditional-blocks --experimental %clang_args -D__CUDA_API_VERSION_INTERNAL -ferror-limit=500

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
// CHECK: #include "hiptensor.h"
#include "hiptensor.h"
// CHECK-NOT: #include "hiptensor.h"

int main() {
  printf("25. cuTensor API to hipTensor API synthetic test\n");

  // CHECK: hiptensorHandle *handle_p = nullptr;
  // CHECK-NEXT: hiptensorHandle_t handle;
  // CHECK-NEXT: const hiptensorHandle_t *handle_c = nullptr;
  // CHECK-NEXT: hiptensorHandle_t *handle2 = nullptr;
  hiptensorHandle *handle_p = nullptr;
  hiptensorHandle_t handle;
  const hiptensorHandle_t *handle_c = nullptr;
  hiptensorHandle_t *handle2 = nullptr;

  //CHECK: hiptensorStatus_t status;
  hiptensorStatus_t status;

  //CHECK: hiptensorTensorDescriptor_t *tensorDescriptor = nullptr;
  //CHECK-NEXT: hiptensorTensorDescriptor_t *descA = nullptr;
  //CHECK-NEXT: hiptensorTensorDescriptor_t *descB = nullptr;
  //CHECK-NEXT: hiptensorTensorDescriptor_t *descC = nullptr;
  //CHECK-NEXT: hiptensorTensorDescriptor_t *descD = nullptr;
  hiptensorTensorDescriptor_t *tensorDescriptor = nullptr;
  hiptensorTensorDescriptor_t *descA = nullptr;
  hiptensorTensorDescriptor_t *descB = nullptr;
  hiptensorTensorDescriptor_t *descC = nullptr;
  hiptensorTensorDescriptor_t *descD = nullptr;

#if CUDA_VERSION >= 8000
  // CHECK: hipDataType dataType;
  hipDataType dataType;
#endif

  // CHECK: hipStream_t stream_t;
  hipStream_t stream_t;

  const uint32_t numModes = 0;
  uint32_t numCachelinesRead = 0;
  const int64_t *extent = nullptr;
  const int64_t *stride = nullptr;
  const uint64_t workspaceSize = 0;
  uint64_t workspaceSize2 = 0;
  const void *alpha = nullptr;
  const void *A = nullptr;
  const int32_t *modeA = nullptr;
  void *B = nullptr;
  const void *B_1 = nullptr;
  const void *beta = nullptr;
  const int32_t *modeB = nullptr;
  const void *C = nullptr;
  const int32_t *modeC = nullptr;
  void *D = nullptr;
  const int32_t *modeD = nullptr;
  void *workspace = nullptr;
  const char *err = nullptr;
  const char *log = nullptr;
  const char *filename = nullptr;
  size_t ver = 0;
  FILE *file = nullptr;
  int32_t level = 0;
  int32_t mask = 0;

#if CUTENSOR_MAJOR >= 2
  // CHECK: hiptensorDataType_t tensorDataType_t;
  // CHECK-NEXT hiptensorDataType_t TENSOR_R_16F = HIPTENSOR_R_16F;
  // CHECK-NEXT hiptensorDataType_t TENSOR_C_16F = HIPTENSOR_C_16F;
  // CHECK-NEXT hiptensorDataType_t TENSOR_R_16BF = HIPTENSOR_R_16BF;
  // CHECK-NEXT hiptensorDataType_t TENSOR_C_16BF = HIPTENSOR_C_16BF;
  // CHECK-NEXT hiptensorDataType_t TENSOR_R_32F = HIPTENSOR_R_32F;
  // CHECK-NEXT hiptensorDataType_t TENSOR_C_32F = HIPTENSOR_C_32F;
  // CHECK-NEXT hiptensorDataType_t TENSOR_R_64F = HIPTENSOR_R_64F;
  // CHECK-NEXT hiptensorDataType_t TENSOR_C_64F = HIPTENSOR_C_64F;
  // CHECK-NEXT hiptensorDataType_t TENSOR_R_8I = HIPTENSOR_R_8I;
  // CHECK-NEXT hiptensorDataType_t TENSOR_R_8U = HIPTENSOR_R_8U;
  // CHECK-NEXT hiptensorDataType_t TENSOR_R_32I = HIPTENSOR_R_32I;
  // CHECK-NEXT hiptensorDataType_t TENSOR_R_32U = HIPTENSOR_R_32U;
  // CHECK-NEXT hiptensorDataType_t TENSOR_R_4I = HIPTENSOR_R_4I;
  // CHECK-NEXT hiptensorDataType_t TENSOR_C_4I = HIPTENSOR_C_4I;
  // CHECK-NEXT hiptensorDataType_t TENSOR_R_4U = HIPTENSOR_R_4U;
  // CHECK-NEXT hiptensorDataType_t TENSOR_C_4U = HIPTENSOR_C_4U;
  // CHECK-NEXT hiptensorDataType_t TENSOR_C_8I = HIPTENSOR_C_8I;
  // CHECK-NEXT hiptensorDataType_t TENSOR_C_8U = HIPTENSOR_C_8U;
  // CHECK-NEXT hiptensorDataType_t TENSOR_R_16I = HIPTENSOR_R_16I;
  // CHECK-NEXT hiptensorDataType_t TENSOR_C_16I = HIPTENSOR_C_16I;
  // CHECK-NEXT hiptensorDataType_t TENSOR_R_16U = HIPTENSOR_R_16U;
  // CHECK-NEXT hiptensorDataType_t TENSOR_C_16U = HIPTENSOR_C_16U;
  // CHECK-NEXT hiptensorDataType_t TENSOR_C_32I = HIPTENSOR_C_32I;
  // CHECK-NEXT hiptensorDataType_t TENSOR_C_32U = HIPTENSOR_C_32U;
  // CHECK-NEXT hiptensorDataType_t TENSOR_R_64I = HIPTENSOR_R_64I;
  // CHECK-NEXT hiptensorDataType_t TENSOR_C_64I = HIPTENSOR_C_64I;
  // CHECK-NEXT hiptensorDataType_t TENSOR_R_64U = HIPTENSOR_R_64U;
  // CHECK-NEXT hiptensorDataType_t TENSOR_C_64U = HIPTENSOR_C_64U;
  hiptensorDataType_t tensorDataType_t;
  hiptensorDataType_t TENSOR_R_16F = HIPTENSOR_R_16F;
  hiptensorDataType_t TENSOR_C_16F = HIPTENSOR_C_16F;
  hiptensorDataType_t TENSOR_R_16BF = HIPTENSOR_R_16BF;
  hiptensorDataType_t TENSOR_C_16BF = HIPTENSOR_C_16BF;
  hiptensorDataType_t TENSOR_R_32F = HIPTENSOR_R_32F;
  hiptensorDataType_t TENSOR_C_32F = HIPTENSOR_C_32F;
  hiptensorDataType_t TENSOR_R_64F = HIPTENSOR_R_64F;
  hiptensorDataType_t TENSOR_C_64F = HIPTENSOR_C_64F;
  hiptensorDataType_t TENSOR_R_8I = HIPTENSOR_R_8I;
  hiptensorDataType_t TENSOR_R_8U = HIPTENSOR_R_8U;
  hiptensorDataType_t TENSOR_R_32I = HIPTENSOR_R_32I;
  hiptensorDataType_t TENSOR_R_32U = HIPTENSOR_R_32U;
  hiptensorDataType_t TENSOR_R_4I = HIPTENSOR_R_4I;
  hiptensorDataType_t TENSOR_C_4I = HIPTENSOR_C_4I;
  hiptensorDataType_t TENSOR_R_4U = HIPTENSOR_R_4U;
  hiptensorDataType_t TENSOR_C_4U = HIPTENSOR_C_4U;
  hiptensorDataType_t TENSOR_C_8I = HIPTENSOR_C_8I;
  hiptensorDataType_t TENSOR_C_8U = HIPTENSOR_C_8U;
  hiptensorDataType_t TENSOR_R_16I = HIPTENSOR_R_16I;
  hiptensorDataType_t TENSOR_C_16I = HIPTENSOR_C_16I;
  hiptensorDataType_t TENSOR_R_16U = HIPTENSOR_R_16U;
  hiptensorDataType_t TENSOR_C_16U = HIPTENSOR_C_16U;
  hiptensorDataType_t TENSOR_C_32I = HIPTENSOR_C_32I;
  hiptensorDataType_t TENSOR_C_32U = HIPTENSOR_C_32U;
  hiptensorDataType_t TENSOR_R_64I = HIPTENSOR_R_64I;
  hiptensorDataType_t TENSOR_C_64I = HIPTENSOR_C_64I;
  hiptensorDataType_t TENSOR_R_64U = HIPTENSOR_R_64U;
  hiptensorDataType_t TENSOR_C_64U = HIPTENSOR_C_64U;

  // CHECK: hiptensorOperationDescriptorAttribute_t tensorOperationDescriptorAttribute_t;
  // CHECK-NEXT hiptensorOperationDescriptorAttribute_t TENSOR_OPERATION_DESCRIPTOR_TAG = HIPTENSOR_OPERATION_DESCRIPTOR_TAG;
  // CHECK-NEXT hiptensorOperationDescriptorAttribute_t TENSOR_OPERATION_DESCRIPTOR_SCALAR_TYPE = HIPTENSOR_OPERATION_DESCRIPTOR_SCALAR_TYPE;
  // CHECK-NEXT hiptensorOperationDescriptorAttribute_t TENSOR_OPERATION_DESCRIPTOR_FLOPS = HIPTENSOR_OPERATION_DESCRIPTOR_FLOPS;
  // CHECK-NEXT hiptensorOperationDescriptorAttribute_t TENSOR_OPERATION_DESCRIPTOR_MOVED_BYTES = HIPTENSOR_OPERATION_DESCRIPTOR_MOVED_BYTES;
  // CHECK-NEXT hiptensorOperationDescriptorAttribute_t TENSOR_OPERATION_DESCRIPTOR_PADDING_LEFT = HIPTENSOR_OPERATION_DESCRIPTOR_PADDING_LEFT;
  // CHECK-NEXT hiptensorOperationDescriptorAttribute_t TENSOR_OPERATION_DESCRIPTOR_PADDING_RIGHT = HIPTENSOR_OPERATION_DESCRIPTOR_PADDING_RIGHT;
  // CHECK-NEXT hiptensorOperationDescriptorAttribute_t TENSOR_OPERATION_DESCRIPTOR_PADDING_VALUE = HIPTENSOR_OPERATION_DESCRIPTOR_PADDING_VALUE;
  hiptensorOperationDescriptorAttribute_t tensorOperationDescriptorAttribute_t;
  hiptensorOperationDescriptorAttribute_t TENSOR_OPERATION_DESCRIPTOR_TAG = HIPTENSOR_OPERATION_DESCRIPTOR_TAG;
  hiptensorOperationDescriptorAttribute_t TENSOR_OPERATION_DESCRIPTOR_SCALAR_TYPE = HIPTENSOR_OPERATION_DESCRIPTOR_SCALAR_TYPE;
  hiptensorOperationDescriptorAttribute_t TENSOR_OPERATION_DESCRIPTOR_FLOPS = HIPTENSOR_OPERATION_DESCRIPTOR_FLOPS;
  hiptensorOperationDescriptorAttribute_t TENSOR_OPERATION_DESCRIPTOR_MOVED_BYTES = HIPTENSOR_OPERATION_DESCRIPTOR_MOVED_BYTES;
  hiptensorOperationDescriptorAttribute_t TENSOR_OPERATION_DESCRIPTOR_PADDING_LEFT = HIPTENSOR_OPERATION_DESCRIPTOR_PADDING_LEFT;
  hiptensorOperationDescriptorAttribute_t TENSOR_OPERATION_DESCRIPTOR_PADDING_RIGHT = HIPTENSOR_OPERATION_DESCRIPTOR_PADDING_RIGHT;
  hiptensorOperationDescriptorAttribute_t TENSOR_OPERATION_DESCRIPTOR_PADDING_VALUE = HIPTENSOR_OPERATION_DESCRIPTOR_PADDING_VALUE;

  // CHECK: hiptensorPlanPreferenceAttribute_t tensorPlanPreferenceAttribute_t;
  // CHECK-NEXT hiptensorPlanPreferenceAttribute_t TENSOR_PLAN_PREFERENCE_AUTOTUNE_MODE = HIPTENSOR_PLAN_PREFERENCE_AUTOTUNE_MODE;
  // CHECK-NEXT hiptensorPlanPreferenceAttribute_t TENSOR_PLAN_PREFERENCE_CACHE_MODE = HIPTENSOR_PLAN_PREFERENCE_CACHE_MODE;
  // CHECK-NEXT hiptensorPlanPreferenceAttribute_t TENSOR_PLAN_PREFERENCE_INCREMENTAL_COUNT = HIPTENSOR_PLAN_PREFERENCE_INCREMENTAL_COUNT;
  // CHECK-NEXT hiptensorPlanPreferenceAttribute_t TENSOR_PLAN_PREFERENCE_ALGO = HIPTENSOR_PLAN_PREFERENCE_ALGO;
  // CHECK-NEXT hiptensorPlanPreferenceAttribute_t TENSOR_PLAN_PREFERENCE_KERNEL_RANK = HIPTENSOR_PLAN_PREFERENCE_KERNEL_RANK;
  // CHECK-NEXT hiptensorPlanPreferenceAttribute_t TENSOR_PLAN_PREFERENCE_JIT = HIPTENSOR_PLAN_PREFERENCE_JIT;
  hiptensorPlanPreferenceAttribute_t tensorPlanPreferenceAttribute_t;
  hiptensorPlanPreferenceAttribute_t TENSOR_PLAN_PREFERENCE_AUTOTUNE_MODE = HIPTENSOR_PLAN_PREFERENCE_AUTOTUNE_MODE;
  hiptensorPlanPreferenceAttribute_t TENSOR_PLAN_PREFERENCE_CACHE_MODE = HIPTENSOR_PLAN_PREFERENCE_CACHE_MODE;
  hiptensorPlanPreferenceAttribute_t TENSOR_PLAN_PREFERENCE_INCREMENTAL_COUNT = HIPTENSOR_PLAN_PREFERENCE_INCREMENTAL_COUNT;
  hiptensorPlanPreferenceAttribute_t TENSOR_PLAN_PREFERENCE_ALGO = HIPTENSOR_PLAN_PREFERENCE_ALGO;
  hiptensorPlanPreferenceAttribute_t TENSOR_PLAN_PREFERENCE_KERNEL_RANK = HIPTENSOR_PLAN_PREFERENCE_KERNEL_RANK;
  hiptensorPlanPreferenceAttribute_t TENSOR_PLAN_PREFERENCE_JIT = HIPTENSOR_PLAN_PREFERENCE_JIT;

  // CHECK: hiptensorJitMode_t tensorJitMode_t;
  // CHECK-NEXT hiptensorJitMode_t TENSOR_JIT_MODE_NONE = HIPTENSOR_JIT_MODE_NONE;
  // CHECK-NEXT hiptensorJitMode_t TENSOR_JIT_MODE_DEFAULT = HIPTENSOR_JIT_MODE_DEFAULT;
  hiptensorJitMode_t tensorJitMode_t;
  hiptensorJitMode_t TENSOR_JIT_MODE_NONE = HIPTENSOR_JIT_MODE_NONE;
  hiptensorJitMode_t TENSOR_JIT_MODE_DEFAULT = HIPTENSOR_JIT_MODE_DEFAULT;

  // CHECK: hiptensorPlanAttribute_t tensorPlanAttribute_t;
  // CHECK-NEXT hiptensorPlanAttribute_t TENSOR_PLAN_REQUIRED_WORKSPACE = HIPTENSOR_PLAN_REQUIRED_WORKSPACE;
  hiptensorPlanAttribute_t tensorPlanAttribute_t;
  hiptensorPlanAttribute_t TENSOR_PLAN_REQUIRED_WORKSPACE = HIPTENSOR_PLAN_REQUIRED_WORKSPACE;

  // CHECK: hiptensorAutotuneMode_t TENSOR_AUTOTUNE_MODE_NONE = HIPTENSOR_AUTOTUNE_MODE_NONE;
  // CHECK-NEXT hiptensorAutotuneMode_t TENSOR_AUTOTUNE_MODE_INCREMENTAL = HIPTENSOR_AUTOTUNE_MODE_INCREMENTAL;
  hiptensorAutotuneMode_t TENSOR_AUTOTUNE_MODE_NONE = HIPTENSOR_AUTOTUNE_MODE_NONE;
  hiptensorAutotuneMode_t TENSOR_AUTOTUNE_MODE_INCREMENTAL = HIPTENSOR_AUTOTUNE_MODE_INCREMENTAL;

  // CHECK: hiptensorPlan *tensorPlan_p = nullptr;
  // CHECK-NEXT: hiptensorPlan_t tensorPlan_t;
  hiptensorPlan *tensorPlan_p = nullptr;
  hiptensorPlan_t tensorPlan_t;

  // CHECK: hiptensorPlanPreference *tensorPlanRef_p = nullptr;
  // CHECK-NEXT: hiptensorPlanPreference_t tensorPlanPreference_t;
  hiptensorPlanPreference *tensorPlanRef_p = nullptr;
  hiptensorPlanPreference_t tensorPlanPreference_t;

  // CHECK: hiptensorOperationDescriptor *tensorOperationDescriptor_p = nullptr;
  // CHECK-NEXT: hiptensorOperationDescriptor_t tensorOperationDescriptor_t;
  hiptensorOperationDescriptor *tensorOperationDescriptor_p = nullptr;
  hiptensorOperationDescriptor_t tensorOperationDescriptor_t;

  // CUDA: hiptensorStatus_t hiptensorContract(const hiptensorHandle_t handle, const hiptensorPlan_t plan, const void* alpha, const void *A, const void *B, const void* beta, const void *C, void *D, void* workspace, uint64_t workspaceSize, hipStream_t stream);
  // HIP: hiptensorStatus_t hiptensorContraction(const hiptensorHandle_t* handle, const hiptensorContractionPlan_t* plan, const void* alpha, const void* A, const void* B, const void* beta, const void* C, void* D, void* workspace, uint64_t workspaceSize, hipStream_t stream);
  // CHECK: status = hiptensorContraction(handle, tensorPlan_t, alpha, A, B_1, beta, C, D, workspace,  workspaceSize2, stream_t);
  status = hiptensorContract(handle, tensorPlan_t, alpha, A, B_1, beta, C, D, workspace, workspaceSize2, stream_t);

  // CUDA: hiptensorStatus_t hiptensorCreate(hiptensorHandle_t* handle);
  // HIP: hiptensorStatus_t hiptensorCreate(hiptensorHandle_t* handle);
  // CHECK: status = hiptensorCreate(&handle);
  status = hiptensorCreate(&handle);

  // CUDA: hiptensorStatus_t hiptensorDestroy(hiptensorHandle_t handle);
  // HIP: hiptensorStatus_t hiptensorDestroy(hiptensorHandle_t handle);
  // CHECK: status = hiptensorDestroy(handle);
  status = hiptensorDestroy(handle);

  // CUDA: hiptensorStatus_t hiptensorHandleResizePlanCache(hiptensorHandle_t handle, const uint32_t numEntries);
  // HIP: hiptensorStatus_t hiptensorHandleResizePlanCache(hiptensorHandle_t handle, const uint32_t numEntries);
  // CHECK: status = hiptensorHandleResizePlanCache(handle, numModes);
  status = hiptensorHandleResizePlanCache(handle, numModes);

  // CUDA: hiptensorStatus_t hiptensorHandleWritePlanCacheToFile(const hiptensorHandle_t handle, const char filename[]);
  // HIP: hiptensorStatus_t hiptensorHandleWritePlanCacheToFile(const hiptensorHandle_t handle, const char filename[]);
  // CHECK: status = hiptensorHandleWritePlanCacheToFile(handle, filename);
  status = hiptensorHandleWritePlanCacheToFile(handle, filename);

  // CUDA: hiptensorStatus_t hiptensorHandleReadPlanCacheFromFile(hiptensorHandle_t handle, const char filename[], uint32_t* numCachelinesRead);
  // HIP: hiptensorStatus_t hiptensorHandleReadPlanCacheFromFile(hiptensorHandle_t handle, const char filename[], uint32_t* numCachelinesRead);
  // CHECK: status = hiptensorHandleReadPlanCacheFromFile(handle, filename, &numCachelinesRead);
  status = hiptensorHandleReadPlanCacheFromFile(handle, filename, &numCachelinesRead);

  // CUDA: hiptensorStatus_t hiptensorWriteKernelCacheToFile(const hiptensorHandle_t handle, const char filename[]);
  // HIP: hiptensorStatus_t hiptensorWriteKernelCacheToFile(const hiptensorHandle_t handle, const char filename[]);
  // CHECK: status = hiptensorWriteKernelCacheToFile(handle, filename);
  status = hiptensorWriteKernelCacheToFile(handle, filename);

  // CUDA: hiptensorStatus_t hiptensorReadKernelCacheFromFile(hiptensorHandle_t handle, const char filename[]);
  // HIP: hiptensorStatus_t hiptensorReadKernelCacheFromFile(hiptensorHandle_t handle, const char filename[]);
  // CHECK: status = hiptensorReadKernelCacheFromFile(handle, filename);
  status = hiptensorReadKernelCacheFromFile(handle, filename);
#endif

#if CUTENSOR_MAJOR >= 1
  // CHECK: hiptensorOperator_t tensorOperator_t;
  // CHECK-NEXT hiptensorOperator_t TENSOR_OP_IDENTITY = HIPTENSOR_OP_IDENTITY;
  // CHECK-NEXT hiptensorOperator_t TENSOR_OP_SQRT = HIPTENSOR_OP_SQRT;
  // CHECK-NEXT hiptensorOperator_t TENSOR_OP_RELU = HIPTENSOR_OP_RELU;
  // CHECK-NEXT hiptensorOperator_t TENSOR_OP_CONJ = HIPTENSOR_OP_CONJ;
  // CHECK-NEXT hiptensorOperator_t TENSOR_OP_RCP = HIPTENSOR_OP_RCP;
  // CHECK-NEXT hiptensorOperator_t TENSOR_OP_SIGMOID = HIPTENSOR_OP_SIGM
  // CHECK-NEXT hiptensorOperator_t TENSOR_OP_TANH = HIPTENSOR_OP_TANH;
  // CHECK-NEXT hiptensorOperator_t TENSOR_OP_EXP = HIPTENSOR_OP_EXP;
  // CHECK-NEXT hiptensorOperator_t TENSOR_OP_LOG = HIPTENSOR_OP_LOG;
  // CHECK-NEXT hiptensorOperator_t TENSOR_OP_ABS = HIPTENSOR_OP_ABS;
  // CHECK-NEXT hiptensorOperator_t TENSOR_OP_NEG = HIPTENSOR_OP_NEG;
  // CHECK-NEXT hiptensorOperator_t TENSOR_OP_SIN = HIPTENSOR_OP_SIN;
  // CHECK-NEXT hiptensorOperator_t TENSOR_OP_COS = HIPTENSOR_OP_COS;
  // CHECK-NEXT hiptensorOperator_t TENSOR_OP_TAN = HIPTENSOR_OP_TAN;
  // CHECK-NEXT hiptensorOperator_t TENSOR_OP_SINH = HIPTENSOR_OP_SINH;
  // CHECK-NEXT hiptensorOperator_t TENSOR_OP_COSH = HIPTENSOR_OP_COSH;
  // CHECK-NEXT hiptensorOperator_t TENSOR_OP_ASIN = HIPTENSOR_OP_ASIN;
  // CHECK-NEXT hiptensorOperator_t TENSOR_OP_ACOS = HIPTENSOR_OP_ACOS;
  // CHECK-NEXT hiptensorOperator_t TENSOR_OP_ATAN = HIPTENSOR_OP_ATAN;
  // CHECK-NEXT hiptensorOperator_t TENSOR_OP_ASINH = HIPTENSOR_OP_ASINH;
  // CHECK-NEXT hiptensorOperator_t TENSOR_OP_ACOSH = HIPTENSOR_OP_ACOSH;
  // CHECK-NEXT hiptensorOperator_t TENSOR_OP_ATANH = HIPTENSOR_OP_ATANH;
  // CHECK-NEXT hiptensorOperator_t TENSOR_OP_CEIL = HIPTENSOR_OP_CEIL;
  // CHECK-NEXT hiptensorOperator_t TENSOR_OP_FLOOR = HIPTENSOR_OP_FLOOR;
  // CHECK-NEXT hiptensorOperator_t TENSOR_OP_ADD = HIPTENSOR_OP_ADD;
  // CHECK-NEXT hiptensorOperator_t TENSOR_OP_MUL = HIPTENSOR_OP_MUL;
  // CHECK-NEXT hiptensorOperator_t TENSOR_OP_MAX = HIPTENSOR_OP_MAX;
  // CHECK-NEXT hiptensorOperator_t TENSOR_OP_MIN = HIPTENSOR_OP_MIN;
  // CHECK-NEXT hiptensorOperator_t TENSOR_OP_UNKNOWN = HIPTENSOR_OP_UNKNOWN;
  hiptensorOperator_t tensorOperator_t;
  hiptensorOperator_t TENSOR_OP_IDENTITY = HIPTENSOR_OP_IDENTITY;
  hiptensorOperator_t TENSOR_OP_SQRT = HIPTENSOR_OP_SQRT;
  hiptensorOperator_t TENSOR_OP_RELU = HIPTENSOR_OP_RELU;
  hiptensorOperator_t TENSOR_OP_CONJ = HIPTENSOR_OP_CONJ;
  hiptensorOperator_t TENSOR_OP_RCP = HIPTENSOR_OP_RCP;
  hiptensorOperator_t TENSOR_OP_SIGMOID = HIPTENSOR_OP_SIGMOID;
  hiptensorOperator_t TENSOR_OP_TANH = HIPTENSOR_OP_TANH;
  hiptensorOperator_t TENSOR_OP_EXP = HIPTENSOR_OP_EXP;
  hiptensorOperator_t TENSOR_OP_LOG = HIPTENSOR_OP_LOG;
  hiptensorOperator_t TENSOR_OP_ABS = HIPTENSOR_OP_ABS;
  hiptensorOperator_t TENSOR_OP_NEG = HIPTENSOR_OP_NEG;
  hiptensorOperator_t TENSOR_OP_SIN = HIPTENSOR_OP_SIN;
  hiptensorOperator_t TENSOR_OP_COS = HIPTENSOR_OP_COS;
  hiptensorOperator_t TENSOR_OP_TAN = HIPTENSOR_OP_TAN;
  hiptensorOperator_t TENSOR_OP_SINH = HIPTENSOR_OP_SINH;
  hiptensorOperator_t TENSOR_OP_COSH = HIPTENSOR_OP_COSH;
  hiptensorOperator_t TENSOR_OP_ASIN = HIPTENSOR_OP_ASIN;
  hiptensorOperator_t TENSOR_OP_ACOS = HIPTENSOR_OP_ACOS;
  hiptensorOperator_t TENSOR_OP_ATAN = HIPTENSOR_OP_ATAN;
  hiptensorOperator_t TENSOR_OP_ASINH = HIPTENSOR_OP_ASINH;
  hiptensorOperator_t TENSOR_OP_ACOSH = HIPTENSOR_OP_ACOSH;
  hiptensorOperator_t TENSOR_OP_ATANH = HIPTENSOR_OP_ATANH;
  hiptensorOperator_t TENSOR_OP_CEIL = HIPTENSOR_OP_CEIL;
  hiptensorOperator_t TENSOR_OP_FLOOR = HIPTENSOR_OP_FLOOR;
  hiptensorOperator_t TENSOR_OP_ADD = HIPTENSOR_OP_ADD;
  hiptensorOperator_t TENSOR_OP_MUL = HIPTENSOR_OP_MUL;
  hiptensorOperator_t TENSOR_OP_MAX = HIPTENSOR_OP_MAX;
  hiptensorOperator_t TENSOR_OP_MIN = HIPTENSOR_OP_MIN;
  hiptensorOperator_t TENSOR_OP_UNKNOWN = HIPTENSOR_OP_UNKNOWN;

  // CHECK: hiptensorStatus_t tensorStatus_t;
  // CHECK-NEXT hiptensorStatus_t TENSOR_STATUS_SUCCESS = HIPTENSOR_STATUS_SUCCESS;
  // CHECK-NEXT hiptensorStatus_t TENSOR_STATUS_NOT_INITIALIZED = HIPTENSOR_STATUS_NOT_INITIALIZED;
  // CHECK-NEXT hiptensorStatus_t TENSOR_STATUS_ALLOC_FAILED = HIPTENSOR_STATUS_ALLOC_FAILED;
  // CHECK-NEXT hiptensorStatus_t TENSOR_STATUS_INVALID_VALUE = HIPTENSOR_STATUS_INVALID_VALUE;
  // CHECK-NEXT hiptensorStatus_t TENSOR_STATUS_ARCH_MISMATCH = HIPTENSOR_STATUS_ARCH_MISMATCH;
  // CHECK-NEXT hiptensorStatus_t TENSOR_STATUS_EXECUTION_FAILED = HIPTENSOR_STATUS_EXECUTION_FAILED;
  // CHECK-NEXT hiptensorStatus_t TENSOR_STATUS_INTERNAL_ERROR = HIPTENSOR_STATUS_INTERNAL_ERROR;
  // CHECK-NEXT hiptensorStatus_t TENSOR_STATUS_NOT_SUPPORTED = HIPTENSOR_STATUS_NOT_SUPPORTED;
  // CHECK-NEXT hiptensorStatus_t TENSOR_STATUS_INSUFFICIENT_WORKSPACE = HIPTENSOR_STATUS_INSUFFICIENT_WORKSPACE;
  // CHECK-NEXT hiptensorStatus_t TENSOR_STATUS_INSUFFICIENT_DRIVER = HIPTENSOR_STATUS_INSUFFICIENT_DRIVER;
  hiptensorStatus_t tensorStatus_t;
  hiptensorStatus_t TENSOR_STATUS_SUCCESS = HIPTENSOR_STATUS_SUCCESS;
  hiptensorStatus_t TENSOR_STATUS_NOT_INITIALIZED = HIPTENSOR_STATUS_NOT_INITIALIZED;
  hiptensorStatus_t TENSOR_STATUS_ALLOC_FAILED = HIPTENSOR_STATUS_ALLOC_FAILED;
  hiptensorStatus_t TENSOR_STATUS_INVALID_VALUE = HIPTENSOR_STATUS_INVALID_VALUE;
  hiptensorStatus_t TENSOR_STATUS_ARCH_MISMATCH = HIPTENSOR_STATUS_ARCH_MISMATCH;
  hiptensorStatus_t TENSOR_STATUS_EXECUTION_FAILED = HIPTENSOR_STATUS_EXECUTION_FAILED;
  hiptensorStatus_t TENSOR_STATUS_INTERNAL_ERROR = HIPTENSOR_STATUS_INTERNAL_ERROR;
  hiptensorStatus_t TENSOR_STATUS_NOT_SUPPORTED = HIPTENSOR_STATUS_NOT_SUPPORTED;
  hiptensorStatus_t TENSOR_STATUS_INSUFFICIENT_WORKSPACE = HIPTENSOR_STATUS_INSUFFICIENT_WORKSPACE;
  hiptensorStatus_t TENSOR_STATUS_INSUFFICIENT_DRIVER = HIPTENSOR_STATUS_INSUFFICIENT_DRIVER;

  // CHECK: hiptensorAlgo_t tensorAlgo_t;
  // CHECK-NEXT hiptensorAlgo_t TENSOR_ALGO_DEFAULT = HIPTENSOR_ALGO_DEFAULT;
  hiptensorAlgo_t tensorAlgo_t;
  hiptensorAlgo_t TENSOR_ALGO_DEFAULT = HIPTENSOR_ALGO_DEFAULT;

  // CHECK: hiptensorWorksizePreference_t tensorWorksizePreference_t;
  // Check-NEXT TENSOR_WORKSPACE_MIN = HIPTENSOR_WORKSPACE_MIN;
  // CHECK-NEXT TENSOR_WORKSPACE_MAX = HIPTENSOR_WORKSPACE_MAX;
  hiptensorWorksizePreference_t tensorWorksizePreference_t;
  hiptensorWorksizePreference_t TENSOR_WORKSPACE_MIN = HIPTENSOR_WORKSPACE_MIN;
  hiptensorWorksizePreference_t TENSOR_WORKSPACE_MAX = HIPTENSOR_WORKSPACE_MAX;

  // CUDA: const char* hiptensorGetErrorString(const hiptensorStatus_t error);
  // HIP: const char* hiptensorGetErrorString(const hiptensorStatus_t error);
  // CHECK: err = hiptensorGetErrorString(status);
  err = hiptensorGetErrorString(status);

  // CUDA: size_t hiptensorGetHiprtVersion();
  // HIP: int hiptensorGetHiprtVersion();
  // CHECK: ver = hiptensorGetHiprtVersion();
  ver = hiptensorGetHiprtVersion();
#endif

#if (CUTENSOR_MAJOR == 1 && CUTENSOR_MINOR >= 4) || CUTENSOR_MAJOR >= 2
  // CHECK: hiptensorAlgo_t TENSOR_ALGO_DEFAULT_PATIENT = HIPTENSOR_ALGO_DEFAULT_PATIENT;
  hiptensorAlgo_t TENSOR_ALGO_DEFAULT_PATIENT = HIPTENSOR_ALGO_DEFAULT_PATIENT;
#endif

#if (CUTENSOR_MAJOR >= 1 && CUTENSOR_MAJOR < 2)
  // CHECK: hiptensorComputeDescriptor_t tensorComputeType_t;
  hiptensorComputeDescriptor_t tensorComputeType_t;

  // CHECK: hiptensorContractionPlan_t tensorPlan2_t;
  hiptensorContractionPlan_t tensorPlan2_t;

#if CUTENSOR_MINOR >= 2
  // CHECK: hiptensorAutotuneMode_t tensorAutotuneMode_t;
  hiptensorAutotuneMode_t tensorAutotuneMode_t;

  // CHECK: hiptensorStatus_t TENSOR_STATUS_IO_ERROR = HIPTENSOR_STATUS_IO_ERROR;
  hiptensorStatus_t TENSOR_STATUS_IO_ERROR = HIPTENSOR_STATUS_IO_ERROR;

  // CHECK hiptensorComputeDescriptor_t TENSOR_COMPUTE_16F = HIPTENSOR_COMPUTE_DESC_16F;
  // CHECK-NEXT hiptensorComputeDescriptor_t TENSOR_COMPUTE_16BF = HIPTENSOR_COMPUTE_DESC_16BF;
  // CHECK-NEXT hiptensorComputeDescriptor_t TENSOR_COMPUTE_32F = HIPTENSOR_COMPUTE_DESC_32F;
  // CHECK-NEXT hiptensorComputeDescriptor_t TENSOR_COMPUTE_64F = HIPTENSOR_COMPUTE_DESC_64F;
  // CHECK-NEXT hiptensorComputeDescriptor_t TENSOR_COMPUTE_8U = HIPTENSOR_COMPUTE_DESC_8U;
  // CHECK-NEXT hiptensorComputeDescriptor_t TENSOR_COMPUTE_8I = HIPTENSOR_COMPUTE_DESC_8I;
  // CHECK-NEXT hiptensorComputeDescriptor_t TENSOR_COMPUTE_32U = HIPTENSOR_COMPUTE_DESC_32U;
  // CHECK-NEXT hiptensorComputeDescriptor_t TENSOR_COMPUTE_32I = HIPTENSOR_COMPUTE_DESC_32I;
  hiptensorComputeDescriptor_t TENSOR_COMPUTE_16F = HIPTENSOR_COMPUTE_DESC_16F;
  hiptensorComputeDescriptor_t TENSOR_COMPUTE_16BF = HIPTENSOR_COMPUTE_DESC_16BF;
  hiptensorComputeDescriptor_t TENSOR_COMPUTE_32F = HIPTENSOR_COMPUTE_DESC_32F;
  hiptensorComputeDescriptor_t TENSOR_COMPUTE_64F = HIPTENSOR_COMPUTE_DESC_64F;
  hiptensorComputeDescriptor_t TENSOR_COMPUTE_8U = HIPTENSOR_COMPUTE_DESC_8U;
  hiptensorComputeDescriptor_t TENSOR_COMPUTE_8I = HIPTENSOR_COMPUTE_DESC_8I;
  hiptensorComputeDescriptor_t TENSOR_COMPUTE_32U = HIPTENSOR_COMPUTE_DESC_32U;
  hiptensorComputeDescriptor_t TENSOR_COMPUTE_32I = HIPTENSOR_COMPUTE_DESC_32I;

  // CHECK: hiptensorCacheMode_t tensorCacheMode_t;
  // CHECK-NEXT hiptensorCacheMode_t TENSOR_CACHE_MODE_NONE = HIPTENSOR_CACHE_MODE_NONE;
  // CHECK-NEXT hiptensorCacheMode_t TENSOR_CACHE_MODE_PEDANTIC = HIPTENSOR_CACHE_MODE_PEDANTIC;
  hiptensorCacheMode_t tensorCacheMode_t;
  hiptensorCacheMode_t TENSOR_CACHE_MODE_NONE = HIPTENSOR_CACHE_MODE_NONE;
  hiptensorCacheMode_t TENSOR_CACHE_MODE_PEDANTIC = HIPTENSOR_CACHE_MODE_PEDANTIC;
#endif

  // CHECK: const hiptensorContractionPlan_t *plan_c = nullptr;
  const hiptensorContractionPlan_t *plan_c = nullptr;

  // CHECK: hiptensorWorksizePreference_t TENSOR_WORKSPACE_DEFAULT = HIPTENSOR_WORKSPACE_DEFAULT;
  hiptensorWorksizePreference_t TENSOR_WORKSPACE_DEFAULT = HIPTENSOR_WORKSPACE_DEFAULT;

#if CUDA_VERSION >= 8000
  // CUDA: hiptensorStatus_t hiptensorInitTensorDescriptor(const hiptensorHandle_t* handle, hiptensorTensorDescriptor_t* desc, const uint32_t numModes, const int64_t extent[], const int64_t stride[], hipDataType dataType, hiptensorOperator_t unaryOp);
  // HIP: hiptensorStatus_t hiptensorInitTensorDescriptor(const hiptensorHandle_t* handle, hiptensorTensorDescriptor_t* desc, const uint32_t numModes, const int64_t lens[], const int64_t strides[], hipDataType dataType, hiptensorOperator_t unaryOp);
  // CHECK: status = hiptensorInitTensorDescriptor(handle_c, tensorDescriptor, numModes, extent, stride, dataType, tensorOperator_t);
  status = hiptensorInitTensorDescriptor(handle_c, tensorDescriptor, numModes, extent, stride, dataType, tensorOperator_t);
#endif

  // CUDA: hiptensorStatus_t hiptensorPermutation(const hiptensorHandle_t* handle, const void* alpha, const void* A, const hiptensorTensorDescriptor_t* descA, const int32_t modeA[], void* B, const hiptensorTensorDescriptor_t* descB, const int32_t modeB[], const hipDataType typeScalar, const hipStream_t stream);
  // HIP: hiptensorStatus_t hiptensorPermutation(const hiptensorHandle_t* handle, const void* alpha, const void* A, const hiptensorTensorDescriptor_t* descA, const int32_t modeA[], void* B, const hiptensorTensorDescriptor_t* descB, const int32_t modeB[], const hipDataType typeScalar, const hipStream_t stream);
  // CHECK: status = hiptensorPermutation(handle_c, alpha, A, descA, modeA, B, descB, modeB, dataType, stream_t);
  status = hiptensorPermutation(handle_c, alpha, A, descA, modeA, B, descB, modeB, dataType, stream_t);

  // CUDA: hiptensorStatus_t hiptensorContraction(const hiptensorHandle_t* handle, const hiptensorContractionPlan_t* plan, const void* alpha, const void* A, const void* B, const void* beta, const void* C, void* D, void *workspace, uint64_t workspaceSize, hipStream_t stream);
  // HIP: hiptensorStatus_t hiptensorContraction(const hiptensorHandle_t* handle, const hiptensorContractionPlan_t* plan, const void* alpha, const void* A, const void* B, const void* beta, const void* C, void* D, void* workspace, uint64_t workspaceSize, hipStream_t stream);
  // CHECK: status = hiptensorContraction(handle_c, plan_c, alpha, A, B_1, beta, C, D, workspace, workspaceSize, stream_t);
  status = hiptensorContraction(handle_c, plan_c, alpha, A, B_1, beta, C, D, workspace, workspaceSize, stream_t);

  // CUDA: hiptensorStatus_t hiptensorReduction(const hiptensorHandle_t* handle, const void* alpha, const void* A, const hiptensorTensorDescriptor_t* descA, const int32_t modeA[], const void* beta, const void* C, const hiptensorTensorDescriptor_t* descC, const int32_t modeC[], void* D, const hiptensorTensorDescriptor_t* descD, const int32_t modeD[], hiptensorOperator_t opReduce, hiptensorComputeDescriptor_t typeCompute, void *workspace, uint64_t workspaceSize, hipStream_t stream);
  // HIP: hiptensorStatus_t hiptensorReduction(const hiptensorHandle_t* handle, const void* alpha, const void* A, const hiptensorTensorDescriptor_t* descA, const int32_t modeA[], const void* beta, const void* C, const hiptensorTensorDescriptor_t* descC, const int32_t modeC[], void* D, const hiptensorTensorDescriptor_t* descD,  const int32_t modeD[], hiptensorOperator_t opReduce, hiptensorComputeType_t typeCompute, void* workspace, uint64_t workspaceSize, hipStream_t stream);
  // CHECK: status = hiptensorReduction(handle_c, alpha, A, descA, modeA, beta, C, descC, modeC, D, descD, modeD, tensorOperator_t, tensorComputeType_t, workspace, workspaceSize2, stream_t);
  status = hiptensorReduction(handle_c, alpha, A, descA, modeA, beta, C, descC, modeC, D, descD, modeD, tensorOperator_t, tensorComputeType_t, workspace, workspaceSize2, stream_t);
#endif

#if (CUTENSOR_MAJOR == 1 && CUTENSOR_MINOR >= 3 && CUTENSOR_PATCH >= 2) || CUTENSOR_MAJOR >= 2
  // CHECK: hiptensorLoggerCallback_t callback;
  hiptensorLoggerCallback_t callback;

  // CUDA: hiptensorStatus_t hiptensorLoggerSetCallback(hiptensorLoggerCallback_t callback);
  // HIP: hiptensorStatus_t hiptensorLoggerSetCallback(hiptensorLoggerCallback_t callback);
  // CHECK: status = hiptensorLoggerSetCallback(callback);
  status = hiptensorLoggerSetCallback(callback);

  // CUDA: hiptensorStatus_t hiptensorLoggerSetFile(FILE* file);
  // HIP: hiptensorStatus_t hiptensorLoggerSetFile(FILE* file);
  // CHECK: status = hiptensorLoggerSetFile(file);
  status = hiptensorLoggerSetFile(file);

  // CUDA: hiptensorStatus_t hiptensorLoggerOpenFile(const char* logFile);
  // HIP: hiptensorStatus_t hiptensorLoggerOpenFile(const char* logFile);
  // CHECK: status = hiptensorLoggerOpenFile(log);
  status = hiptensorLoggerOpenFile(log);

  // CUDA: hiptensorStatus_t hiptensorLoggerSetLevel(int32_t level);
  // HIP: hiptensorStatus_t hiptensorLoggerSetLevel(hiptensorLogLevel_t level);
  // CHECK: status = hiptensorLoggerSetLevel(level);
  status = hiptensorLoggerSetLevel(level);

  // CUDA: hiptensorStatus_t hiptensorLoggerSetMask(int32_t mask);
  // HIP: hiptensorStatus_t hiptensorLoggerSetMask(int32_t mask);
  // CHECK: status = hiptensorLoggerSetMask(mask);
  status = hiptensorLoggerSetMask(mask);

  // CUDA: hiptensorStatus_t hiptensorLoggerForceDisable();
  // HIP: hiptensorStatus_t hiptensorLoggerForceDisable();
  // CHECK: status = hiptensorLoggerForceDisable();
  status = hiptensorLoggerForceDisable();
#endif

  return 0;
}
